#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "config.h"

#include <math.h>

template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
    renderCUDA(
        const int numGaussians,
        const float* __restrict__ opacity,
        const float2* __restrict__ means,
        const float2* __restrict__ stds,
        const float* __restrict__ rhos,
        const float* __restrict__ colors,
        const float* __restrict__ grad_output,
        const int sH, int sW,
        const float scaleFactor,
        const float rasterRatio,
        float* __restrict__ dL_dopacity,
        float2* __restrict__ dL_dmeans,
        float2* __restrict__ dL_dstds,
        float* __restrict__ dL_drhos,
        float* __restrict__ dL_dcolors
    )
{
    // Make this method to be per gaussian instead of per pixel

    // Get all Gaussian Parameters and necessary variables for Eq. 1 and 2
    int gaussianIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gaussianIdx >= numGaussians) return;

    float alfa = opacity[gaussianIdx];
    float stdX = stds[gaussianIdx].x;
    float stdY = stds[gaussianIdx].y;
    float stdXY = stdX * stdY;
    float stdX2 = stdX * stdX;
    float stdY2 = stdY * stdY;
    float meanX = means[gaussianIdx].x;
    float meanY = means[gaussianIdx].y;
    float rho = rhos[gaussianIdx];
    float beta = 1 - rho * rho;
    float betaRoot = sqrt(beta);
    float exp1 = -1 / (2 * beta);

    // Initialize gaussian gradient values
    float grad_opacity = 0;
    float grad_meanX = 0;
    float grad_meanY = 0;
    float grad_stdX = 0;
    float grad_stdY = 0;
    float grad_rho = 0;

    float rsH = rasterRatio * sH;
    float rsW = rasterRatio * sW;

    // Iterate through all pixels of the image
    for (int x = 0; x < sH; x++) {
        for (int y = 0; y < sW; y++) {
            // Get pixel coordinates and check if pixel is within the Gaussian influence
            float deltaX = (x - meanX);
            float deltaY = (y - meanY);
            if (fabs(deltaX) >= rsH || fabs(deltaY) >= rsW) continue;

            // Finish computing Eq. 1
            float deltaX2 = deltaX * deltaX;
            float deltaY2 = deltaY * deltaY;
            float deltaXY = deltaX * deltaY;
            float exp2 = deltaX2 / stdX2 + deltaY2 / stdY2 - 2 * rho * deltaXY / stdXY;
            float f = 1 / (2 * M_PI * stdXY * betaRoot) * exp(exp1 * exp2);

            // Now compute gradients
            for (int c = 0; c < CHANNELS; c++) {
                int idx = x * sW * CHANNELS + y * CHANNELS + c;
                float grad = grad_output[idx];

                if (grad != 0) {
                    float color = colors[gaussianIdx * CHANNELS + c];

                    // Opacity grad
                    grad_opacity += grad * f * color;
                    // Mean X grad
                    float dL_df = grad * alfa * color;
                    float df_dmeanx = f * exp1 * ((-2 * deltaX / stdX2) + (2 * rho * deltaY / stdXY));
                    grad_meanX += dL_df * df_dmeanx;
                    // Mean Y grad
                    float df_dmeany = f * exp1 * ((-2 * deltaY / stdY2) + (2 * rho * deltaX / stdXY));
                    grad_meanY += dL_df * df_dmeany;
                    // Std X grad
                    float df_dstdx = -f / stdX + f * exp1 * (-2 * deltaX2 / stdX2 / stdX + 2 * rho * deltaXY / stdXY / stdX);
                    grad_stdX += dL_df * df_dstdx;
                    // Std Y grad
                    float df_dstdy = -f / stdY + f * exp1 * (-2 * deltaY2 / stdY2 / stdY + 2 * rho * deltaXY / stdXY / stdY);
                    grad_stdY += dL_df * df_dstdy;
                    // Rho grad
                    float df_drho = f * ((rho / beta) - (rho * exp2 / (beta * beta)) - (exp1 * 2 * deltaXY / stdXY));
                    grad_rho += dL_df * df_drho;
                    // Color grad
                    dL_dcolors[gaussianIdx * CHANNELS + c] += grad * alfa * f;
                }
            }
        }
    }
    dL_dopacity[gaussianIdx] = grad_opacity;
    dL_dmeans[gaussianIdx].x = grad_meanX;
    dL_dmeans[gaussianIdx].y = grad_meanY;
    dL_dstds[gaussianIdx].x = grad_stdX;
    dL_dstds[gaussianIdx].y = grad_stdY;
    dL_drhos[gaussianIdx] = grad_rho;
}

void BACKWARD::render(
    const dim3 grid, dim3 block,
    const int numGaussians,
    const float* __restrict__ opacity,
    const float2* __restrict__ means,
    const float2* __restrict__ stds,
    const float* __restrict__ rhos,
    const float* __restrict__ colors,
    const float* __restrict__ grad_output,
    const int sH, int sW,
    const float scaleFactor,
    const float rasterRatio,
    float* __restrict__ dL_dopacity,
    float2* __restrict__ dL_dmeans,
    float2* __restrict__ dL_dstds,
    float* __restrict__ dL_drhos,
    float* __restrict__ dL_dcolors
)
{
    renderCUDA<NUM_CHANNELS><<<grid, block>>>(
        numGaussians,
        opacity,
        means,
        stds,
        rhos,
        colors,
        grad_output,
        sH, sW,
        scaleFactor,
        rasterRatio,
        dL_dopacity,
        dL_dmeans,
        dL_dstds,
        dL_drhos,
        dL_dcolors
    );
}

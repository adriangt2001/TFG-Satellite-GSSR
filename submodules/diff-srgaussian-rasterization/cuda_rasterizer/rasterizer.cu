/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "config.h"
#include "forward.h"
#include "backward.h"

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(bool debug)
{
    dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    dim3 block(BLOCK_X, BLOCK_Y, 1);

    // Let each tile blend its range of Gaussians independently in parallel
    const float *feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
    CHECK_CUDA(FORWARD::render(), debug)

    return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(bool debug)
{
    const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    const dim3 block(BLOCK_X, BLOCK_Y, 1);

    // Compute loss gradients
    CHECK_CUDA(BACKWARD::render(), debug);
}